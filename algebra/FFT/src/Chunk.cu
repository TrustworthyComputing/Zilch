
#include <hip/hip_runtime.h>
//#include "Chunk.cuh"
//#include <bitset>
//#include <iostream>
//#include <cstdio>
//
//using namespace std;
//namespace FFF{
//__constant__ idx_t p_Mod[max_nonzero_coefs_in_mod];
//__constant__ idx_t p_ModLen;
//__constant__ Element element_mul;
//__constant__ Chunk c;
//
//
//__device__ void a_chunkToNormal(Chunk *d_a, Elements_Chunk *d_b, idx_t idx)
//{
//	cell_t ans = 0;
//	idx_t element_idx = idx & andMask(Chunk::log_elements_in_chunk);
//	idx_t cell_idx = idx >> Chunk::log_elements_in_chunk;
//	for(unsigned int i = cell_idx<<Element::log_bits_in_cell ; i < ((cell_idx+1)<<Element::log_bits_in_cell); ++i)
//		ans^=(((cell_t)(((d_a->v[i])>>(element_idx))&1))<<(i-(cell_idx<<Element::log_bits_in_cell)));
//	d_b->e[element_idx].c[cell_idx]=ans;
//}
//__global__ void k_chunkToNormal(Chunk *d_a,Elements_Chunk *d_b , len_t len)
//{
//	const unsigned int threads_in_chunk = Chunk::elements_in_chunk * Element::element_len;
//	 __shared__ Chunk input[max_block_size / threads_in_chunk];
//	idx_t idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if(idx >= len*threads_in_chunk)
//		return;
//	idx_t chunkIdx = (idx) / (Element::element_len*Chunk::elements_in_chunk);
//	idx_t in_chunkIdx = (idx  & (Element::element_len * Chunk::elements_in_chunk - 1));
//	idx_t chunks_in_block = blockDim.x / Chunk::cells_in_chunk;
//	idx_t inBlockChunkIdx = chunkIdx & (threads_in_chunk-1);
//	for(unsigned int i = 0 ; i < sizeof(cell_t)/sizeof(chunk_cell_t) ; ++i){
//		input[inBlockChunkIdx].v[in_chunkIdx + i*threads_in_chunk] = d_a[chunkIdx].v[in_chunkIdx+i*threads_in_chunk];
//	}
//	a_chunkToNormal(&(input[inBlockChunkIdx]), &(d_b[chunkIdx]),in_chunkIdx);
//}
//__host__	void Chunk::chunkToNormal(Chunk(*h_a), Elements_Chunk(*h_b), len_t len, bool copy)
//{
//	//Declare device variables
//	Chunk (*d_a);
//	Elements_Chunk (*d_b);
//
//	const unsigned int num_element = len*elements_in_chunk;
//	const unsigned int threads = Element::element_len * num_element;
//
//	//Define Block  and Grid Size.
//	dim3 blockSize(max_block_size,1,1);
//	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
//	if(copy){
//	//Allocate Memory on GPU. (global)
//		cudaMalloc(&d_a,sizeof(Chunk)*len);
//		cudaMalloc(&d_b,sizeof(Elements_Chunk)*len);
//
//	//Copy memory to GPU.
//		cudaMemcpy(d_a,h_a,sizeof(Chunk)*len,cudaMemcpyHostToDevice);
//	} else {
//		d_a = h_a;
//		d_b = h_b;
//	}
//
//	//Launch Kernel
//	k_chunkToNormal<<<gridSize,blockSize>>>(d_a,d_b,len);
//	if(copy){
//	//Copy results back to memory
//		cudaMemcpy(h_b,d_b,sizeof(Elements_Chunk)*len,cudaMemcpyDeviceToHost);
//
//	//Free allocated memory.
//		cudaFree(d_a);
//		cudaFree(d_b);
//	}
//}
//__device__ void a_normalToChunk(Elements_Chunk *d_a, Chunk *d_b, idx_t idx)
//{
//	chunk_cell_t ans = 0;
//	idx_t cell_idx = idx>>Element::log_bits_in_cell;
//	for(unsigned int i = 0 ; i < Chunk::elements_in_chunk ; ++i)
//		ans^=((((d_a->e[i].c[cell_idx])>>(idx& andMask(Element::log_bits_in_cell)))&1)<<i);
//	d_b->v[idx]=ans;
//}
//__global__		void k_normalToChunk(Elements_Chunk *d_a,Chunk *d_b , len_t len)
//{
//	idx_t idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if(idx >= (len<<Chunk::log_cells_in_chunk))
//		return;
//	idx_t chunkIdx = (idx) >> Chunk::log_cells_in_chunk;
//	idx_t in_chunkIdx = (idx  & andMask(Chunk::log_cells_in_chunk));
//	a_normalToChunk(&(d_a[chunkIdx]),&(d_b[chunkIdx]),in_chunkIdx);
//}
//__host__	void Chunk::normalToChunk(Elements_Chunk(*h_a), Chunk (*h_b), len_t len,bool copy)
//{
//
//	//Declare device variables
//	Elements_Chunk (*d_a);
//	Chunk (*d_b);
//
//	const unsigned int threads = len<<Chunk::log_cells_in_chunk;
//
//	//Define Block  and Grid Size.
//	dim3 blockSize(max_block_size,1,1);
//	dim3 gridSize(sizeCiel(threads,max_block_size),1,1);
//
//	//Allocate Memory on GPU. (global)
//	if(copy){
//		cudaMalloc(&d_a,sizeof(Elements_Chunk)*len);
//		cudaMalloc(&d_b,sizeof(Chunk)*len);
//
//	//Copy memory to GPU.
//		cudaMemcpy(d_a,h_a,sizeof(Elements_Chunk)*len,cudaMemcpyHostToDevice);
//	} else{
//		d_a = h_a;
//		d_b = h_b;
//	}
//	//Launch Kernel
//	k_normalToChunk<<<gridSize,blockSize>>>(d_a,d_b,len);
//
//	//Copy results back to memory
//	if(copy){
//		cudaMemcpy(h_b,d_b,sizeof(Chunk)*len,cudaMemcpyDeviceToHost);
//		//Free allocated memory.
//		cudaFree(d_a);
//		cudaFree(d_b);
//	}
//
//}
//__host__ 	void Chunk::setMod(){
//	cudaMemcpyToSymbol(p_Mod,&(Element::irr_poly_index[ord>>log_warp_size]),sizeof(idx_t)*max_nonzero_coefs_in_mod);
//	cudaMemcpyToSymbol(p_ModLen,&(Element::mod_len[ord>>log_warp_size]),sizeof(idx_t));
//}
//__device__ void Chunk::chunk_reduce_xor(Chunk *a, Chunk *c_bottom, Chunk*c_top, idx_t idx)
//{
//	chunk_cell_t ans=c_bottom->v[idx];
//	unsigned int temp_idx;
//	for(idx_t i = 0 ; i < p_ModLen ; ++i)
//	{
//		for(idx_t j = 0 ; j < p_ModLen ; ++j)
//		{
//			temp_idx = idx+(ord<<1)-p_Mod[i]-p_Mod[j];
//			if(temp_idx >= (ord<<1)-p_Mod[j] && temp_idx < (ord<<1))
//				ans^=c_top->v[temp_idx-ord];
//		}
//	}
//	a->v[idx]^=ans;
//}
//__device__ void Chunk::chunk_xor(Chunk *a, Chunk* b, idx_t idx){
//	a->v[idx]^=b->v[idx];
//}
//__device__ void Chunk::chunk_reduce_xor(Chunk *a, Chunk *c_bottom, idx_t idx,Chunk* to_xor ,int shift)
//{
//	unsigned int k = p_ModLen;
//	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
//		for(unsigned int j = 0 ; j+1 < k ; ++j)
//		{
//			c_bottom->v[(ord>>1)+idx+i+p_Mod[j]]^=c_bottom->v[(ord>>1)+ord+idx+i];
//		}
//	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
//		for(unsigned int j = 0 ; (j+1) < k ; ++j)
//		{
//			c_bottom->v[idx+i+p_Mod[j]]^=c_bottom->v[ord+idx+i];
//		}
//		for(unsigned int i = 0 ; i < ord ; i+=warp_size){
//			to_xor->v[idx+i]^=(c_bottom->v[idx+i]>>shift);
//		}
//}
//
//__device__ void Chunk::chunk_reduce(Chunk *a, Chunk *c_bottom, idx_t idx)
//{
//	unsigned int k = p_ModLen;
//	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
//		for(unsigned int j = 0 ; j+1 < k ; ++j)
//		{
//			c_bottom->v[(ord>>1)+idx+i+p_Mod[j]]^=c_bottom->v[(ord>>1)+ord+idx+i];
//		}
//	for(unsigned int i = 0 ; i < (ord>>1); i+=warp_size)
//		for(unsigned int j = 0 ; (j+1) < k ; ++j)
//		{
//			c_bottom->v[idx+i+p_Mod[j]]^=c_bottom->v[ord+idx+i];
//		}
//		for(unsigned int i = 0 ; i < ord ; i+=warp_size){
//			a->v[idx+i]=c_bottom->v[idx+i];
//		}
//}
//__device__ void Chunk::chunkClmul(Chunk (*a),  Element (*e), idx_t idx, Chunk (*c))
//{
//	chunk_cell_t my_ans[2][(ord>>(log_warp_size))]={0};
//	for(unsigned int k = 0 ; k < ord ; ++k)
//	{
//		if(EXTRACT_BIT(e->c,k))
//			for(unsigned int t = 0 ; t <  (ord>>log_warp_size); ++t)
//			{
//				int b = (k>(idx+warp_size*t));
//				my_ans[b][t]^=a->v[idx+warp_size*t+(b<<log_ord)-k];
//			}
//	}
//	for(unsigned int i = 0 ; i < (ord>>log_warp_size); ++i)
//	{
//		c->v[idx+i*warp_size] = my_ans[0][i];
//		c->v[ord+idx+i*warp_size] = my_ans[1][i];
//	}
//}
//__device__ void Chunk::aux_k_clmul(Chunk *a, Element* e, len_t len,Chunk* c_shared)
//{
//
//	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
//	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
//	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
//	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
//	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
//	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size)
//		my_shared_chunk->v[in_chunk_idx+i]=a[chunk_idx].v[in_chunk_idx+i];
//	Chunk::chunkClmul(my_shared_chunk,e,in_chunk_idx,my_shared_chunk);
//	Chunk::chunk_reduce(a+chunk_idx,my_shared_chunk,in_chunk_idx);
//}
//__global__ void k_clmul(Chunk *a,Element *e,len_t len )
//{
//	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
//	__shared__ Chunk c_shared[shared_len<<1];
//	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
//	if(idx>=(len<<Chunk::log_threads_in_chunk))
//		return;
//	Chunk::aux_k_clmul(a,e,len,c_shared);
//}
//__host__	void Chunk::mul(Chunk (*h_a),Element (*h_e),len_t len, Chunk (*h_res)){
//
//#ifdef __MEASURE
//	cudaEvent_t start,stop;
//	float time;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//#endif
//	//Declare device variables
//	Chunk 	(*d_a);
//	Element (*d_e);
//
//	//Define Block  and Grid Size.
//	dim3 blockSize(max_block_size,1,1);
//	dim3 gridSize(sizeCiel(len<<Chunk::log_threads_in_chunk,max_block_size),1,1);
//
//	//Allocate Memory on GPU. (global)
//	cudaMalloc(&d_a,sizeof(Chunk)*len);
//	cudaMalloc(&d_e,sizeof(Element));
//
//	//Copy memory to GPU.
//	cudaMemcpy(d_a,h_a,sizeof(Chunk)*len,cudaMemcpyHostToDevice);
//	cudaMemcpy(d_e,h_e,sizeof(Element),cudaMemcpyHostToDevice);
//
//	//Set Mod
//	setMod();
////	setElementMul(h_e);
//	//Launch Kernel
//#ifdef __MEASURE
//	cudaEventRecord(start,0);
//#endif
//	k_clmul<<<gridSize,blockSize>>>(d_a,d_e,len);
//#ifdef __MEASURE
//	cudaEventRecord(stop,0);
//#endif
//
//	//Copy results to host
//	cudaMemcpy(h_res,d_a,sizeof(Chunk)*len,cudaMemcpyDeviceToHost);
//	//Free allocated memory.
//	cudaFree(d_a);
//	cudaFree(d_e);
//#ifdef __MEASURE
//	cudaEventElapsedTime(&time,start,stop);
//	printf("Time for the mul: %f ms on %d chunks \n",time,len);
//#endif
//}
//
//__global__ void k_add(Chunk (*a), Chunk (*b), len_t l)
//{
//	unsigned int idx = threadIdx.x+blockIdx.x*blockDim.x;
//	if(idx>=l*Chunk::cells_in_chunk)
//		return;
//	((chunk_cell_t*)a)[idx]^=((chunk_cell_t*)b)[idx];
//}
//__host__ void Chunk::add(Chunk (*h_a),Chunk (*h_b),len_t len)
//{
//
//	//Declare device variables
//	Chunk (*d_a);
//	Chunk (*d_b);
//
//	//Define Block  and Grid Size.
//	dim3 blockSize(max_block_size,1,1);
//	dim3 gridSize(sizeCiel(max_block_size,len),1,1);
//
//	//Allocate Memory on GPU. (global)
//	cudaMalloc(&d_a,sizeof(Chunk)*len);
//	cudaMalloc(&d_b,sizeof(Chunk)*len);
//
//	//Copy memory to GPU.
//	cudaMemcpy(d_a,h_a,sizeof(Chunk)*len,cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b,h_b,sizeof(Chunk)*len,cudaMemcpyHostToDevice);
//
//	//Launch Kernel
//	k_add<<<gridSize,blockSize>>>(d_a,d_b,len);
//
//	//Copy results to CPU memory
//	cudaMemcpy(h_a,d_a,sizeof(Chunk)*len,cudaMemcpyDeviceToHost);
//
//	//Free allocated memory.
//	cudaFree(d_a);
//	cudaFree(d_b);
//}
//__host__ void Chunk::print() const {
//	for(unsigned int i = 0 ; i < cells_in_chunk ; ++i){
//		cout << bitset<bits_in_byte*sizeof(chunk_cell_t)>(this->v[i])<<endl;
//	}
//}
//__host__	void Elements_Chunk::print() const{
//		for(unsigned int i = 0 ; i < elements_in_elements_chunk ; ++i){
//			Element::printElement(this->e[i]);
//			cout<<endl;
//		}
//	}
////Mul chunk by another chunk
//__device__ void Chunk::clmul_by_chunk(Chunk& a, Chunk& e, idx_t idx, Chunk* c){
//	chunk_cell_t my_ans[2][(ord>>(log_warp_size))]={0};
//	for(unsigned int k = 0 ; k < ord ; ++k)
//		for(unsigned int t = 0 ; t <  (ord>>log_warp_size); ++t)
//		{
//			int b = (k>(idx+warp_size*t));
//			my_ans[b][t]^=a.v[idx+warp_size*t+(b<<log_ord)-k]&e.v[k];
//		}
//	for(unsigned int i = 0 ; i < (ord>>log_warp_size); ++i)
//	{
//		c->v[idx+i*warp_size] = my_ans[0][i];
//		c->v[ord+idx+i*warp_size] = my_ans[1][i];
//	}
//}
////Mul chunk by another chunk
//__global__ void k_mul_chunk(Chunk* cs, Chunk* c, len_t cs_len)
//{
//	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
//	__shared__ Chunk c_shared[shared_len<<1];
//	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
//	if(idx>=(cs_len<<Chunk::log_threads_in_chunk))
//		return;
//	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
//	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
//	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
//	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
//	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
//		my_shared_chunk->v[in_chunk_idx+i]=cs[chunk_idx].v[in_chunk_idx+i];
//		my_shared_chunk[1].v[in_chunk_idx+i]=c->v[in_chunk_idx+i];
//	}
//	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
//	Chunk::chunk_reduce(cs+chunk_idx,my_shared_chunk,in_chunk_idx);
//}
//__global__ void k_mul_chunk_xor(Chunk* cs, Chunk* c, len_t cs_len,Chunk* to_xor, int shift = 0)
//{
//	const idx_t shared_len = max_block_size>>Chunk::log_threads_in_chunk;
//	__shared__ Chunk c_shared[shared_len<<1];
//	idx_t idx = blockDim.x*blockIdx.x+threadIdx.x;
//	if(idx>=(cs_len<<Chunk::log_threads_in_chunk))
//		return;
//	const idx_t chunk_idx = (idx >> Chunk::log_threads_in_chunk);
//	const idx_t in_chunk_idx = idx & andMask(Chunk::log_threads_in_chunk);
//	const idx_t shared_chunk_idx = ((idx & andMask(log_max_block_size)) >> (Chunk::log_threads_in_chunk));
//	Chunk* my_shared_chunk = c_shared+(shared_chunk_idx<<1);
//	for(unsigned int i =0 ; i < Chunk::ord ;i+=warp_size){
//		my_shared_chunk->v[in_chunk_idx+i]=cs[chunk_idx].v[in_chunk_idx+i];
//		my_shared_chunk[1].v[in_chunk_idx+i]=c->v[in_chunk_idx+i];
//	}
//	Chunk::clmul_by_chunk(my_shared_chunk[0],my_shared_chunk[1],in_chunk_idx,my_shared_chunk);
//	Chunk::chunk_reduce_xor(cs+chunk_idx,my_shared_chunk,in_chunk_idx,to_xor,shift);
//}
////Mul a chunk by a chunk
//void Chunk::chunk_mul(Chunk (* h_a), Chunk (*h_b) , len_t len, Chunk (*h_res), bool copy, bool do_xor, int shift){
//#ifdef __MEASURE
//	cudaEvent_t start,stop;
//	float time;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//#endif
//	//Declare device variables
//	Chunk 	(*d_a);
//	Chunk   (*d_b);
//
//	//Define Block  and Grid Size.
//	dim3 blockSize(max_block_size,1,1);
//	dim3 gridSize(sizeCiel(len<<Chunk::log_threads_in_chunk,max_block_size),1,1);
//
//	if(copy){
//	//Allocate Memory on GPU. (global)
//		cudaMalloc(&d_a,sizeof(Chunk)*len);
//		cudaMalloc(&d_b,sizeof(Chunk));
//	//Copy memory to GPU.
//		cudaMemcpy(d_a,h_a,sizeof(Chunk)*len,cudaMemcpyHostToDevice);
//		cudaMemcpy(d_b,h_b,sizeof(Chunk),cudaMemcpyHostToDevice);
//	} else {
//		d_a = h_a;
//		d_b = h_b;
//	}
//
//
//	//Set Mod
//	setMod();
////	setElementMul(h_e);
//	//Launch Kernel
//#ifdef __MEASURE
//	cudaEventRecord(start,0);
//#endif
//	if(do_xor)
//		k_mul_chunk_xor<<<gridSize,blockSize>>>(d_a,d_b,len,d_a,shift);
//	else
//		k_mul_chunk<<<gridSize,blockSize>>>(d_a,d_b,len);
//#ifdef __MEASURE
//	cudaEventRecord(stop,0);
//#endif
//
//	if(copy){
//	//Copy results to host
//		cudaMemcpy(h_res,d_a,sizeof(Chunk)*len,cudaMemcpyDeviceToHost);
//	//Free allocated memory.
//		cudaFree(d_a);
//		cudaFree(d_b);
//	}
//#ifdef __MEASURE
//	cudaEventElapsedTime(&time,start,stop);
//	printf("Time for the mul: %f ms on %d chunks \n",time,len);
//#endif
//}
//}
